#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__global__ void init_arr(int *arr, int size) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		arr[tid] = 0;
	}
}

__global__  void kernel_calc_hist(int *arr,int *global_hist, int numOfElements) {
    	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ int shared_hist[RANGE + 1];

    	// Initialize shared histogram values to zero 
        shared_hist[tid] = 0;

	 __syncthreads();

	// Each thread updates shared histogram by the data value.
	// atomicAdd used to avoid race condition between threads in the same block
	if (tid < numOfElements)
	{
		atomicAdd(&shared_hist[arr[tid]], 1);
	}

	__syncthreads();

	
	// Each thread updates global histogram
	if (shared_hist[tid + 1] != 0) {
		global_hist[tid + 1] += shared_hist[tid + 1];
	}
}


int cuda_task(int* histogram, int* data, int numOfElements) {
	//printf("hello world:  %d %d    |   ", numOfElements, data[1]);

	hipError_t err = hipSuccess;

	size_t size = numOfElements * sizeof(int);

	int *d_data;
	int *d_hist;
	int *temp_hist;

	// Allocate space on host for saving histogram value after kernel_calc_hist
	temp_hist = (int*)calloc(RANGE + 1, sizeof(int));

	// Allocate spcae on device for d_hist
	err = hipMalloc((void**)&d_hist, (RANGE + 1) * sizeof(int));
	if (err != hipSuccess) {
    	    fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	// allocate space on device for inputs data
    	err = hipMalloc((void **)&d_data, size);
    	if (err != hipSuccess) {
    	    fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}


    	int threadsPerBlock = RANGE;
    	int blocksPerGrid =(numOfElements + threadsPerBlock - 1) / threadsPerBlock;

	// Initialize device histogram values to zero
	init_arr<<<1, RANGE + 1>>>(d_hist, RANGE + 1);
	err = hipDeviceSynchronize();

	// Calaulate histogram for each block
    	kernel_calc_hist<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_hist, numOfElements);
    	err = hipGetLastError();
    	if (err != hipSuccess) {
    	    	fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	size = (RANGE + 1) * sizeof(int);

	// Copy d_hist from device to host
	err = hipMemcpy(temp_hist, d_hist, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	// Update final histogram
	for (int i=0; i < RANGE + 1; i++)
	{
		histogram[i] += temp_hist[i];
	}


	// Free allocated memory on GPU
	if (hipFree(d_hist) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	if (hipFree(d_data) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	// Free allocated memory on host
	free(temp_hist);

    return 0;
}

